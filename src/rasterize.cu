#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya
 * @date      2012-2015
 * @copyright University of Pennsylvania & STUDENT
 */

#include "rasterize.h"
//<seqan / parallel.h>
#include <thrust/random.h>
#include <cmath>
#include <vector>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include "rasterizeTools.h"
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/type_ptr.hpp>
#define DEG2RAD  PI/180.f
#define Tess 0
#define Blending 0
struct VertexIn {
	glm::vec3 pos;
	glm::vec3 nor;
	glm::vec3 col;
	// TODO (optional) add other vertex attributes (e.g. texture coordinates)
};
struct VertexOut {
	// TODO
	glm::vec3 pos;
	glm::vec3 nor;
	glm::vec3 col;
};
struct Triangle {
	VertexOut v[3];
};
struct Fragment {
	int dis;
	glm::vec3 color;
	glm::vec3 normal;
	glm::vec3 pos;
	glm::vec3 subcolor[4];
	int subdis[4];
};
int N = 0;
int M = 0;
int mat = 0;
int dev = 0;
static int width = 0;
static int height = 0;
static int *dev_bufIdx = NULL;
static VertexIn *dev_bufVertex = NULL;
static VertexOut *dev_vsOutput = NULL;
static Triangle *dev_primitives = NULL;
static Fragment *dev_depthbuffer = NULL;
static Fragment *dev_fmInput = NULL;
static Fragment *dev_fmOutput = NULL;
static glm::vec3 *dev_framebuffer = NULL;
static int bufIdxSize = 0;
static int vertCount = 0;

__host__ __device__ inline unsigned int utilhash(unsigned int a) {
	a = (a + 0x7ed55d16) + (a << 12);
	a = (a ^ 0xc761c23c) ^ (a >> 19);
	a = (a + 0x165667b1) + (a << 5);
	a = (a + 0xd3a2646c) ^ (a << 9);
	a = (a + 0xfd7046c5) + (a << 3);
	a = (a ^ 0xb55a4f09) ^ (a >> 16);
	return a;
}
__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}
/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__ void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * w);

	if (x < w && y < h) {
		glm::vec3 color;
		color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
		color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
		color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}
__global__ void cleanDepth(Fragment* dev_depthbuffer, Fragment* dev_fmInput, int w, int h)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * w);
	//float t = INFINITY;
	if (x < w && y < h)
	{
		dev_depthbuffer[index].color = glm::vec3(1, 0, 0);
		dev_depthbuffer[index].dis = INFINITY;
		dev_depthbuffer[index].normal = glm::vec3(0, 1, 0);
		dev_fmInput[index].normal = glm::vec3(0, 1, 0);
		dev_fmInput[index].dis = INFINITY;
		dev_fmInput[index].color = glm::vec3(1, 1, 1);
		dev_fmInput[index].normal = glm::vec3(0, 1, 0);
	}
}
// Writes fragment colors to the framebuffer
__global__ void render(int w, int h, Fragment *depthbuffer, glm::vec3 *framebuffer) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * w);

	if (x < w && y < h) {
		framebuffer[index] = depthbuffer[index].color;
	}
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
	width = w;
	height = h;
	hipFree(dev_depthbuffer);
	hipMalloc(&dev_depthbuffer, width * height * sizeof(Fragment));
	hipMemset(dev_depthbuffer, 0, width * height * sizeof(Fragment));
	hipFree(dev_framebuffer);
	hipMalloc(&dev_framebuffer, width * height * sizeof(glm::vec3));
	hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));

	hipFree(dev_fmInput);
	hipMalloc(&dev_fmInput, 4 * width * height * sizeof(Fragment));
	hipMemset(dev_fmInput, 0, 4 * width * height * sizeof(Fragment));

	hipFree(dev_fmOutput);
	hipMalloc(&dev_fmOutput, width * height * sizeof(Fragment));
	hipMemset(dev_fmOutput, 0, width * height * sizeof(Fragment));
	checkCUDAError("rasterizeInit");
}

/**
 * Set all of the buffers necessary for rasterization.
 */

void rasterizeSetBuffers(
	int _bufIdxSize, int *bufIdx,
	int _vertCount, float *bufPos, float *bufNor, float *bufCol, bool resselation) {
	//********************
	resselation = Tess;
	//********************
	bufIdxSize = _bufIdxSize;
	vertCount = _vertCount;

	hipFree(dev_bufIdx);
	hipMalloc(&dev_bufIdx, bufIdxSize * sizeof(int));
	hipMemcpy(dev_bufIdx, bufIdx, bufIdxSize * sizeof(int), hipMemcpyHostToDevice);


	VertexIn *bufVertex = new VertexIn[_vertCount];
	float maxv = -1.f;

	for (int i = 0; i < vertCount; i++) {
		int j = i * 3;
		bufVertex[i].pos = glm::vec3(bufPos[j + 0], bufPos[j + 1], bufPos[j + 2]);
		bufVertex[i].nor = glm::vec3(bufNor[j + 0], bufNor[j + 1], bufNor[j + 2]);
		bufVertex[i].col = glm::vec3(bufCol[j + 0], bufCol[j + 1], bufCol[j + 2]);
		//***********check here....*******//
		float temp = std::max(bufVertex[i].pos.x, std::max(bufVertex[i].pos.y, bufVertex[i].pos.y));
		if (temp>maxv){ maxv = temp; }
	}
	N = (int)maxv + 1;
	hipFree(dev_bufVertex);
	hipMalloc(&dev_bufVertex, vertCount * sizeof(VertexIn));
	hipMemcpy(dev_bufVertex, bufVertex, vertCount * sizeof(VertexIn), hipMemcpyHostToDevice);

	hipFree(dev_vsOutput);
	hipMalloc(&dev_vsOutput, vertCount * sizeof(VertexOut));

	if (!resselation)
	{
		hipFree(dev_primitives);
		hipMalloc(&dev_primitives, vertCount / 3 * sizeof(Triangle));
		hipMemset(dev_primitives, 0, vertCount / 3 * sizeof(Triangle));
		checkCUDAError("rasterizeSetBuffers");
	}
	else
	{
		hipFree(dev_primitives);
		hipMalloc(&dev_primitives, vertCount / 3 * 4 * sizeof(Triangle));
		hipMemset(dev_primitives, 0, vertCount / 3 * 4 * sizeof(Triangle));
		checkCUDAError("rasterizeSetBuffers");
	}

}


__global__ void vertexShader(VertexIn *dev_bufVertex, VertexOut *dev_vsOutput, int vertexCount, glm::mat4 ViewProj){

	int id = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (id < vertexCount){
		//simple orthordox projection 
		//dev_vsOutput[id].pos = dev_bufVertex[id].pos;
		//dev_vsOutput[id].nor = dev_bufVertex[id].nor;

		dev_vsOutput[id].pos = multiplyMV(ViewProj, glm::vec4(dev_bufVertex[id].pos, 1));
		dev_vsOutput[id].nor = multiplyMV(ViewProj, glm::vec4(dev_bufVertex[id].nor, 0));
		dev_vsOutput[id].nor = glm::normalize(dev_vsOutput[id].nor);
		dev_vsOutput[id].col = glm::vec3(1, 0, 0);
		//dev_vsOutput[id].col = dev_bufVertex[id].col;
		//interpolate the normal:smooth normal color??
	}

}
__global__ void PrimitiveAssembly(VertexOut *dev_vsOutput, Triangle * dev_primitives, int verCount)
{
	int id = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (id < verCount / 3){
		dev_primitives[id].v[0].pos = dev_vsOutput[3 * id].pos;//012,345,678
		dev_primitives[id].v[1].pos = dev_vsOutput[3 * id + 1].pos;
		dev_primitives[id].v[2].pos = dev_vsOutput[3 * id + 2].pos;

		dev_primitives[id].v[0].nor = dev_vsOutput[3 * id].nor;//012,345,678
		dev_primitives[id].v[1].nor = dev_vsOutput[3 * id + 1].nor;
		dev_primitives[id].v[2].nor = dev_vsOutput[3 * id + 2].nor;

		dev_primitives[id].v[0].col = dev_vsOutput[3 * id].col;//012,345,678
		dev_primitives[id].v[1].col = dev_vsOutput[3 * id + 1].col;
		dev_primitives[id].v[2].col = dev_vsOutput[3 * id + 2].col;
	}
}

__host__ __device__  bool fequal(float a, float b){
	if (a > b - 0.000001&&a < b + 0.000001){ return true; }
	else return false;
}

__device__ int _atomicMin(int *addr, int val)
{
	int old = *addr, assumed;
	if (old <= val) return old;
	do{
		assumed = old;
		old = atomicCAS(addr, assumed, val);
	} while (old != assumed);
	return old;
}
/*{
	int id = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (id < vertexcount / 3.f)
	{
	glm::vec3 tri[3];
	for (int i = 0; i < 3; i++){
	tri[i] = dev_primitives[id].v[i].pos;
	tri[i].x += N;
	tri[i].y += N;
	tri[i].z += N;
	tri[i].x *= w / (float)(2.f*N);
	tri[i].y *= h / (float)(2.f*N);
	tri[i].z *= w / (float)(2.f*N);
	//because the image is cube anyway...I think multiply should have better result than devide...
	}
	AABB aabb;
	aabb = getAABBForTriangle(tri);

	for (int i = aabb.min.x - 1; i < aabb.max.x + 1; i += 0.5){
	for (int j = aabb.min.y - 1; j < aabb.max.y + 1; j += 0.5){
	if (tri[0].x > w || tri[0].x < 0 || tri[0].y>h || tri[0].x < 0)
	{
	//color[i*w + j].color = glm::vec3(0, 0, 0);//black
	}	//anti-aliansing..multisampling the patern 4 sample every pixel
	glm::vec2 point(i, j);

	glm::vec3 baryc = calculateBarycentricCoordinate(tri, point);
	if (isBarycentricCoordInBounds(baryc))
	{
	int intdepth = getZAtCoordinate(baryc, tri);
	int dis;
	_atomicMin(&dis, intdepth);
	if (intdepth == dis){
	dev_fmInput[i*w + j].subcolor[k] = dev_primitives[id].v[0].nor;
	}
	dev_fmInput[i*w + j].pos = dev_primitives[id].v[0].pos;
	dev_fmInput[i*w + j].normal = dev_primitives[id].v[0].nor;
	}
	}
	}
	//else //pixel have more than 1 color
	//{
	/*	glm::vec3 baryc_p[4];
	int intdepth_s[4];
	for (int p = 0; p < 4; p++)
	{
	baryc_p[p] = calculateBarycentricCoordinate(tri, random_point[p]);
	if (isBarycentricCoordInBounds(baryc_p[p])){
	intdepth_s[p] = getZAtCoordinate(baryc_p[p], tri);
	_atomicMin(&dev_fmInput[i*w + j].subdis[p], intdepth_s[p]);
	if (intdepth_s[p] == dev_fmInput[i*w + j].subdis[p]){
	dev_fmInput[i*w + j].subcolor[p] = dev_primitives[id].v[0].nor;;
	}
	}
	}
	dev_fmInput[i*w + j].pos = dev_primitives[id].v[0].pos;
	dev_fmInput[i*w + j].normal = dev_primitives[id].v[0].nor;
	//	}
	}
	}
	}
	}
	//dev_primitives, dev_fmInput*4, dev_fmOutput
	/*__global__ void rasterization(Triangle * dev_primitives, Fragment *dev_fmInput, int vertexcount, int w, int h, int N)
	{
	int id = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (id < vertexcount / 3.f)
	{
	//potimized boundingbox;
	glm::vec3 tri[3];
	for (int i = 0; i < 3; i++){//(-1,1)+1*w/2
	//(-10,10)+10*w/20
	tri[i] = dev_primitives[id].v[i].pos;
	tri[i].x += N;
	tri[i].y += N;
	tri[i].z += N;
	tri[i].x *= w / (float)(2.f*N);
	tri[i].y *= h / (float)(2.f*N);
	tri[i].z *= w / (float)(2.f*N);
	//because the image is cube anyway...I think multiply should have better result than devide...
	}
	AABB aabb;
	aabb = getAABBForTriangle(tri);

	for (int i = aabb.min.x - 1; i < aabb.max.x + 1; i++){
	for (int j = aabb.min.y - 1; j < aabb.max.y + 1; j++){
	if (tri[0].x > w || tri[0].x < 0 || tri[0].y>h || tri[0].x < 0)
	{
	//color[i*w + j].color = glm::vec3(0, 0, 0);//black
	}	//anti-aliansing..multisampling the patern 4 sample every pixel
	glm::vec2 point(i + 0.5, j + 0.5);
	thrust::default_random_engine rngx = makeSeededRandomEngine(i, id, 1);
	thrust::default_random_engine rngy = makeSeededRandomEngine(j, id, 1);
	thrust::uniform_real_distribution<float> u1(0, 0.5);
	thrust::uniform_real_distribution<float> u2(0.5, 0.999);
	glm::vec2 random_point[4];
	int number = 0;
	//random_point[0].x = i  + u1(rngx);//-1,1
	//random_point[0].y = j  + u1(rngy);

	//random_point[1].x = i + u2(rngx);//-1,-1
	//random_point[1].y = j + u1(rngy);

	//random_point[2].x = i + u1(rngx);//1,1
	//random_point[2].y = j + u2(rngy);

	//random_point[3].x = i + u2(rngx);//i+0+0.22,i+
	//random_point[3].y = j + u2(rngy);
	random_point[0].x = i + 0.25;//-1,1
	random_point[0].y = j + 0.25;

	random_point[1].x = i + 0.25;//-1,-1
	random_point[1].y = j + 0.75;

	random_point[2].x = i + 0.75;//1,1
	random_point[2].y = j + 0.25;

	random_point[3].x = i + 0.75;//i+0+0.22,i+
	random_point[3].y = j + 0.75;
	for (int t = 0; t < 4;t++){
	glm::vec3 baryc_sub = calculateBarycentricCoordinate(tri, random_point[t]);
	if (isBarycentricCoordInBounds(baryc_sub))
	{
	number++;
	}
	}
	/*	if (number == 4)//all in
	{
	glm::vec3 baryc = calculateBarycentricCoordinate(tri, point);
	if (isBarycentricCoordInBounds(baryc)){
	int intdepth = getZAtCoordinate(baryc, tri);
	_atomicMin(&(dev_fmInput[i*w + j].dis), intdepth);
	if (intdepth == dev_fmInput[i*w + j].dis){
	for (int k = 0; k < 4; k++){
	dev_fmInput[i*w + j].subcolor[k] = dev_primitives[id].v[0].nor;
	}
	dev_fmInput[i*w + j].pos= dev_primitives[id].v[0].pos;
	dev_fmInput[i*w + j].normal = dev_primitives[id].v[0].nor;
	}
	}
	}*/
//else //pixel have more than 1 color
//{
/*	glm::vec3 baryc_p[4];
	int intdepth_s[4];
	for (int p = 0; p < 4; p++)
	{
	baryc_p[p] = calculateBarycentricCoordinate(tri, random_point[p]);
	if (isBarycentricCoordInBounds(baryc_p[p])){
	intdepth_s[p] = getZAtCoordinate(baryc_p[p], tri);
	_atomicMin(&dev_fmInput[i*w + j].subdis[p], intdepth_s[p]);
	if (intdepth_s[p] == dev_fmInput[i*w + j].subdis[p]){
	dev_fmInput[i*w + j].subcolor[p] = dev_primitives[id].v[0].nor;;
	}
	}
	}
	dev_fmInput[i*w + j].pos = dev_primitives[id].v[0].pos;
	dev_fmInput[i*w + j].normal = dev_primitives[id].v[0].nor;
	//	}
	}
	}
	}
	}*/

/*__global__ void rasterization(Triangle * dev_primitives, Fragment *dev_fmInput, int vertexcount, int w, int h, int N)
{
	int id = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (id < vertexcount / 3.f)
	{
		//potimized boundingbox; 
		glm::vec3 tri[3];
		for (int i = 0; i < 3; i++){//(-1,1)+1*w/2 
			//(-10,10)+10*w/20 
			tri[i] = dev_primitives[id].v[i].pos;
			tri[i].x += N;
			tri[i].y += N;
			tri[i].z += N;
			tri[i].x *= w / (float)(2.f*N);
			tri[i].y *= h / (float)(2.f*N);
			tri[i].z *= w / (float)(2.f*N);
			//because the image is cube anyway...I think multiply should have better result than devide... 

		}
		AABB aabb;
		aabb = getAABBForTriangle(tri);
		for (int i = aabb.min.x - 1; i < aabb.max.x + 1; i++){
			for (int j = aabb.min.y - 1; j < aabb.max.y + 1; j++){
				glm::vec2 point(i, j);
				glm::vec3 baryc = calculateBarycentricCoordinate(tri, point);
				//simple clip.. 
				if (tri[0].x > w || tri[0].x < 0 || tri[0].y>h || tri[0].x < 0)continue;
				if (isBarycentricCoordInBounds(baryc)){
					//these three normal should be the same since they are on the same face (checked) 
					int intdepth = (int)getZAtCoordinate(baryc, tri);
					//atomicMin(int* address, int val)
					//reads word old located at the address, computes the minimum of old and val, 
					//and stores the result back to memory at the same address. returns old
					atomicMin(&dev_fmInput[i*w + j].dis, intdepth);
					if (dev_fmInput[i*w + j].dis == intdepth){

						dev_fmInput[i*w + j].color = dev_primitives[id].v[0].col;
						dev_fmInput[i*w + j].normal = dev_primitives[id].v[0].nor;
						dev_fmInput[i*w + j].pos = (dev_primitives[id].v[0].pos + dev_primitives[id].v[1].pos + dev_primitives[id].v[2].pos) / 3.f;
					}
				}

			}

		}
	}
}*/
__global__ void rasterization(Triangle * dev_primitives, Fragment *dev_fmInput, int vertexcount, int w, int h, int N)
{
	int id = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (id < vertexcount / 3.f)
	{
		//potimized boundingbox; 
		glm::vec3 tri[3];
		for (int i = 0; i < 3; i++){//(-1,1)+1*w/2 
			//(-10,10)+10*w/20 
			tri[i] = dev_primitives[id].v[i].pos;
			tri[i].x += N;
			tri[i].y += N;
			tri[i].z += N;
			tri[i].x *= w / (float)(2.f*N);
			tri[i].y *= h / (float)(2.f*N);
			tri[i].z *= w / (float)(2.f*N);
			//because the image is cube anyway...I think multiply should have better result than devide... 

		}
		AABB aabb;
		aabb = getAABBForTriangle(tri);
		for (int i = aabb.min.x - 1; i < aabb.max.x + 1; i++){
			for (int j = aabb.min.y - 1; j < aabb.max.y + 1; j++){
				glm::vec2 point(i, j);
				glm::vec3 baryc = calculateBarycentricCoordinate(tri, point);
				//random sample anti-aliansing 1-sample..
				thrust::default_random_engine rngx = makeSeededRandomEngine(i, id, 1);
				thrust::default_random_engine rngy = makeSeededRandomEngine(j, id, 1);
				thrust::uniform_real_distribution<float> u1(0, 1);
				thrust::uniform_real_distribution<float> u2(0.5, 0.999);
				//simple clip.. 
				point =glm::vec2(i + u1(rngx), j + u1(rngy));
				if (tri[0].x > w || tri[0].x < 0 || tri[0].y>h || tri[0].x < 0)continue;
				if (isBarycentricCoordInBounds(baryc)){
					//these three normal should be the same since they are on the same face (checked) 
					int intdepth = (int)getZAtCoordinate(baryc, tri);
					//atomicMin(int* address, int val)
					//reads word old located at the address, computes the minimum of old and val, 
					//and stores the result back to memory at the same address. returns old
					atomicMin(&dev_fmInput[i*w + j].dis, intdepth);
					if (dev_fmInput[i*w + j].dis == intdepth){
						dev_fmInput[i*w + j].color = dev_primitives[id].v[0].col;
						dev_fmInput[i*w + j].normal = dev_primitives[id].v[0].nor;
						dev_fmInput[i*w + j].pos = (dev_primitives[id].v[0].pos + dev_primitives[id].v[1].pos + dev_primitives[id].v[2].pos) / 3.f;
					}
				}

			}

		}
	}
}
__global__ void Tesselation(bool active, VertexOut *dev_vertin, Triangle *dev_triout, int vercount)
{

	int id = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (active&&id < vercount / 3.f)
	{
		int tessel_number = 3;
		glm::vec3 tri[3];
		tri[0] = dev_vertin[3 * id].pos;
		tri[1] = dev_vertin[3 * id + 1].pos;
		tri[2] = dev_vertin[3 * id + 2].pos;
		//default tesselation,generate 4 triangles automativaly
		glm::vec3 vnew[3];
		vnew[0] = (tri[0] + tri[1]) / 2.f;
		vnew[1] = (tri[0] + tri[2]) / 2.f;
		vnew[2] = (tri[2] + tri[1]) / 2.f;

		dev_triout[4 * id].v[0].pos = tri[0];
		dev_triout[4 * id].v[1].pos = vnew[0];
		dev_triout[4 * id].v[2].pos = vnew[1];

		dev_triout[4 * id + 1].v[0].pos = vnew[0];
		dev_triout[4 * id + 1].v[1].pos = tri[1];
		dev_triout[4 * id + 1].v[2].pos = vnew[2];

		dev_triout[4 * id + 2].v[0].pos = vnew[0];
		dev_triout[4 * id + 2].v[1].pos = vnew[2];
		dev_triout[4 * id + 2].v[2].pos = vnew[1];

		dev_triout[4 * id + 3].v[0].pos = vnew[1];
		dev_triout[4 * id + 3].v[1].pos = vnew[2];
		dev_triout[4 * id + 3].v[2].pos = tri[2];
		/*for (int i = 0; i < 4; i++){
			for (int j = 0; j < 3; j++)
			{
			dev_triout[4 * id + i].v[j].nor = dev_vertin[3 * id].nor;
			}
			}*/
		//in order to check :change the normal a little
		for (int i = 0; i < 3; i++){
			{
				dev_triout[4 * id].v[i].nor = glm::normalize(dev_vertin[3 * id].nor + glm::vec3(0.3, 0, 0));
				dev_triout[4 * id + 1].v[i].nor = glm::normalize(dev_vertin[3 * id].nor + glm::vec3(0, 0.3, 0));
				dev_triout[4 * id + 2].v[i].nor = glm::normalize(dev_vertin[3 * id].nor + glm::vec3(0, 0, 0));
				dev_triout[4 * id + 3].v[i].nor = glm::normalize(dev_vertin[3 * id].nor + glm::vec3(0, 0, 0.3));
			}
		}
	}

}
/* scan_line:brute force
glm::vec3 tri[3];
for (int i = 0; i < 3; i++){
tri[i] = dev_primitives[id].v[i].pos;
tri[i].x += 1;
tri[i].y += 1;
tri[i].x *= w / 2.f;
tri[i].y *= h / 2.f;
}
for (int i = 0; i < w; i++){
for (int j = 0; j < h; j++){
glm::vec2 point(i, j);
glm::vec3 baryc = calculateBarycentricCoordinate(tri, point);
if (isBarycentricCoordInBounds(baryc)){
dev_fmInput[i*w + j].color = glm::vec3(1, 0, 0);
}
}*/

glm::vec3 SetLight()
{
	glm::vec3 light_pos = glm::vec3(2, 1, 2);

	return light_pos;
}
//blin phong
/*__global__ void antialiansing(Triangle *dev_in,Triangle *dev_out,int trianglecount)
{
int id = (blockIdx.x * blockDim.x) + threadIdx.x;
if (id < trianglecount)
{

}
}*/
//input output depthbuffer

__global__ void	fragmentShading(Fragment *dev_fmInput, Fragment *dev_fmOutput, int w, int h, glm::vec3 light_pos, glm::vec3 camera_pos, bool defaultbackground)
{
	int id = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (id < w*h){
		__syncthreads();
		///glm::vec3 ccc = (dev_fmInput[id].subcolor[0] + dev_fmInput[id].subcolor[1] + dev_fmInput[id].subcolor[2] + dev_fmInput[id].subcolor[3]) / 4.f;
		glm::vec3 ccc = dev_fmInput[id].color;
		float specular_power = 100;
		glm::vec3 specular_color = glm::vec3(1, 1, 1);//dev_fmInput[id].color;
		glm::vec3 lightray = light_pos - dev_fmInput[id].pos;
		glm::vec3 inray = camera_pos - dev_fmInput[id].pos;
		glm::vec3 H = glm::normalize(inray) + glm::normalize(lightray);
		H = glm::vec3(H.x / 2.0, H.y / 2.0, H.z / 2.0);
		float hdot = glm::dot(H, dev_fmInput[id].normal);
		float x = pow(hdot, specular_power);
		if (x < 0)x = 0.f;
		glm::vec3 spec = x*specular_color;

		glm::vec3 Lambert = glm::vec3(1, 1, 1);
		glm::vec3 Ambient = ccc;
		float diffuse = glm::clamp(glm::dot(dev_fmInput[id].normal, glm::normalize(lightray)), 0.0f, 1.0f);
		Lambert *= diffuse;

		glm::vec3 phong_color = 0.5f*spec + 0.4f*Lambert + 0.1f*Ambient;//where is ambient light?
		phong_color = glm::clamp(phong_color, 0.f, 1.f);

		//dev_fmOutput[id].color = phong_color;
		//blending
		//DestinationColor.rgb = (SourceColor.rgb * One) + (DestinationColor.rgb * (1 - SourceColor.a));
		if (Blending){
			if (defaultbackground)
			{
				glm::vec3 background = glm::vec3(0, 0, 1);
				float default_a = 0.8;
				dev_fmOutput[id].color = phong_color + (background * (1 - default_a));
			}
			else
			{
				float depth = dev_fmInput[id].dis;
				if (depth > 0) {
					dev_fmOutput[id].color = glm::vec3(0.8, 0.8, 0.8);
				}
				else dev_fmOutput[id].color = (-depth)* phong_color + (1 + depth)*glm::vec3(0.8, 0.8, 0.8);
			}
		}
		else dev_fmOutput[id].color = phong_color;
	}
}


/*
 * Perform rasterization.
 */
void RotateAboutRight(float deg, glm::vec3 &ref, const glm::vec3 right, const glm::vec3 eye)
{
	deg *= DEG2RAD;
	glm::mat4 rotation = glm::rotate(glm::mat4(1.0f), deg, right);
	ref = ref - eye;
	ref = glm::vec3(rotation * glm::vec4(ref, 1));
	ref = ref + eye;

}
void TranslateAlongRight(float amt, glm::vec3 &ref, const glm::vec3 right, glm::vec3 &eye)
{
	glm::vec3 translation = right * amt;
	eye += translation;
	ref += translation;
}
void RotateAboutUp(float deg, glm::vec3 &ref, const glm::vec3 right, const glm::vec3 eye, const glm::vec3 up)
{
	deg *= DEG2RAD;
	glm::mat4 rotation = glm::rotate(glm::mat4(1.0f), deg, up);
	ref = ref - eye;
	ref = glm::vec3(rotation * glm::vec4(ref, 1));
	ref = ref + eye;
}
void TranslateAlongLook(float amt, const glm::vec3 look, glm::vec3 &eye, glm::vec3 & ref)
{
	glm::vec3 translation = look * amt;
	eye += translation;
	ref += translation;
}
void TranslateAlongUp(float amt, glm::vec3 &eye, glm::vec3 & ref, const glm::vec3 up)
{
	glm::vec3 translation = up * amt;
	eye += translation;
	ref += translation;
}
glm::mat4 camera(float x_trans_amount, float y_trans_amount, float up_angle_amount, float right_angle_amount, glm::vec3 &camerapos)
{
	glm::vec3 eye = glm::vec3(3, 0, 3);
	glm::vec3 up = glm::vec3(0, 1, 0);
	glm::vec3 ref = glm::vec3(0, 0, 0);
	camerapos = eye;

	float near_clip = 1.0f;
	float far_clip = 1000.f;
	float width = 800;
	float height = 800;
	float aspect = (float)width / (float)height;
	float fovy = 45.f;
	glm::vec3 world_up = glm::vec3(0, 1, 0);
	glm::vec3 look = glm::normalize(ref - eye);
	glm::vec3 right = glm::normalize(glm::cross(look, world_up));
	RotateAboutRight(right_angle_amount, ref, right, eye);
	RotateAboutUp(up_angle_amount, ref, right, eye, up);
	TranslateAlongRight(x_trans_amount, ref, right, eye);
	TranslateAlongUp(y_trans_amount, eye, ref, up);

	glm::mat4 viewMatrix = glm::lookAt(eye, ref, up);
	glm::mat4 projectionMatrix = glm::perspective(fovy, aspect, near_clip, far_clip);//fovy,aspect, zNear, zFar;

	glm::mat4 getViewProj = projectionMatrix*viewMatrix;
	return getViewProj;

}

void rasterize(uchar4 *pbo, float amt_x, float amt_y, float up_a, float right_a)
{
	int sideLength2d = 8;
	dim3 blockSize2d(sideLength2d, sideLength2d);
	dim3 blockCount2d((width - 1) / blockSize2d.x + 1,
		(height - 1) / blockSize2d.y + 1);
	//key_test:
	//std::cout << "ss " << amt_x << "and " << amt_y << std::endl;
	//std::cout << "dd" << up_a << "and " << right_a << std::endl;

	//step1.vertex shading
	int blockSize1d = 256;
	int blockCount1d = (vertCount + blockSize1d - 1) / blockSize1d;

	int image_blockSize1d = 256;
	int image_blockCount1d = (width*height + image_blockSize1d - 1) / image_blockSize1d;
	glm::vec3 camera_pos = glm::vec3(0);
	glm::vec3 light_pos = SetLight();
	glm::mat4 getViewProj = camera(amt_x,amt_y, up_a,right_a, camera_pos);
	//glm::mat4 getViewProj = glm::mat4(1);
	//clean depth buffer
	cleanDepth << < image_blockCount1d, image_blockSize1d >> >(dev_depthbuffer, dev_fmInput, width, height);
	checkCUDAError("clean");
	vertexShader << <blockCount1d, blockSize1d >> >(dev_bufVertex, dev_vsOutput, vertCount, getViewProj);
	checkCUDAError("vertexShader");
	//step2.primitive assembly
	int blockCount1d_tri;
	bool tesselation = Tess;
	if (!tesselation)
	{
		//vertexnumber: vertcount,triangle number:vertcount/3.0
		blockCount1d_tri = blockCount1d / 3 + 1;
		PrimitiveAssembly << < blockCount1d_tri, blockSize1d >> >(dev_vsOutput, dev_primitives, vertCount);
		checkCUDAError("PrimitiveAssembly");
		rasterization << < blockCount1d_tri, blockSize1d >> >(dev_primitives, dev_fmInput, vertCount, width, height, N);
		checkCUDAError("rasterization");
	}
	else
	{
		blockCount1d_tri = blockCount1d / 3 * 4 + 1;
		//vertexnumber: vertcount*12,triangle number:vertcount*12/3.0
		Tesselation << <blockCount1d_tri, blockSize1d >> >(1, dev_vsOutput, dev_primitives, vertCount);
		checkCUDAError("Tesselation");
		rasterization << < blockCount1d_tri, blockSize1d >> >(dev_primitives, dev_fmInput, vertCount * 4, width, height, N);
		checkCUDAError("rasterization");
	}
	//blin-phong+blending
	fragmentShading << <image_blockCount1d, image_blockSize1d >> >(dev_fmInput, dev_depthbuffer, width, height, light_pos, camera_pos, 1);
	checkCUDAError("shading");
	//blending << <image_blockCount1d, image_blockSize1d >> >(dev_fmOutput, dev_depthbuffer, N, 1);
	checkCUDAError("blending");
	render << <blockCount2d, blockSize2d >> >(width, height, dev_depthbuffer, dev_framebuffer);
	sendImageToPBO << <blockCount2d, blockSize2d >> >(pbo, width, height, dev_framebuffer);
	checkCUDAError("sendToPBO");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {
	hipFree(dev_bufIdx);
	dev_bufIdx = NULL;

	hipFree(dev_bufVertex);
	dev_bufVertex = NULL;

	hipFree(dev_primitives);
	dev_primitives = NULL;

	hipFree(dev_vsOutput);
	dev_fmInput = NULL;
	hipFree(dev_fmInput);
	dev_fmInput = NULL;

	hipFree(dev_depthbuffer);
	dev_depthbuffer = NULL;

	hipFree(dev_framebuffer);
	dev_framebuffer = NULL;

	checkCUDAError("rasterizeFree");
}
